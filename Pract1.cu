#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include ""
#include <hip/hip_runtime.h>
#include <math.h>

#define N 10000000

// ядро
__global__ void add(int* a, int* b, int* c) {
    *c = *a + *b;
}
__global__ void copyData(float* input, float* output)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N)
    {
        output[index] = input[index];
    }
}
__global__ void calculatePi(double* result)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        double x = (double)idx / N;
        double y = sqrtf(1 - x * x); // Формула окружности x^2 + y^2 = r^2

        result[idx] = y / N;
    }
}

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
//b.Напишите программу, которая выводит на экран результат сложения двух чисел.-----------------------------------------------------------------------------------
    int a, b, c;

    // переменные на GPU
    int* dev_a, * dev_b, * dev_c;
    int size = sizeof(int); //размерность
    // выделяем память на GPU
    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);
    // инициализация переменных
    a = 2;
    b = 4;
    // копирование информации с CPU на GPU
    hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice
    );
    hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice
    );
    // вызов ядра
    add << < 1, 1 >> > (dev_a, dev_b, dev_c);
    // копирование результата работы ядра с GPU на CPU
        hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);
    // вывод информации
    
    printf("b) %d + %d = %d\n\n", a, b, c);
    // очищение памяти на GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

//c. Напишите программу для определения следующих параметров видеокарты с поддержкой технологии CUDA---------------------------------------------------------------------------
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);//определение параметров GPU с номером 0
        printf("c) Device name : %s\n", deviceProp.name);
        printf("Total global memory : %d MB\n",
            deviceProp.totalGlobalMem / 1024 / 1024);
        printf("Shared memory per block : %d\n",
            deviceProp.sharedMemPerBlock);
        printf("Registers per block : %d\n",
            deviceProp.regsPerBlock);
        printf("Warp size : %d\n", deviceProp.warpSize);
        printf("Memory pitch : %d\n", deviceProp.memPitch);
        printf("Max threads per block : %d\n",
            deviceProp.maxThreadsPerBlock);
        printf("Max threads dimensions : x = %d, y = %d, z = % d\n", deviceProp.maxThreadsDim[0],
            deviceProp.maxThreadsDim[1],
            deviceProp.maxThreadsDim[2]);
        printf("Max grid size: x = %d, y = %d, z = %d\n",
            deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
            deviceProp.maxGridSize[2]);
        printf("Clock rate: %d\n", deviceProp.clockRate);
        printf("Total constant memory: %d\n",
            deviceProp.totalConstMem);
        printf("Compute capability: %d.%d\n",
            deviceProp.major, deviceProp.minor);
        printf("Texture alignment: %d\n",
            deviceProp.textureAlignment);
        printf("Device overlap: %d\n",
            deviceProp.deviceOverlap);
        printf("Multiprocessor count: %d\n",
            deviceProp.multiProcessorCount);
        printf("Kernel execution timeout enabled: %s\n\n",
            deviceProp.kernelExecTimeoutEnabled ? "true" :
            "false");
        scanf("");
    }

//d. Измерьте скорость копирования данных (ГБ/сек) между CPU и GPU.-------------------------------------------------------------------------------------
    //#define N 10000000

    float* h_input = new float[N];
    float* h_output = new float[N];
    float* d_input, * d_output;

    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        h_input[i] = i;
    }

    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    // инициализируем события
    hipEvent_t start, stop;
    float elapsedTime;
    // создаем события
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // запись события
    hipEventRecord(start, 0);
    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);
    copyData << <numBlocks, blockSize >> > (d_input, d_output);
    // вызов ядра
    hipEventRecord(stop, 0);
    // ожидание завершения работы ядра
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    // вывод информации 
    printf("d) Time spent executing by the GPU: %.2f millseconds\n", elapsedTime);
    double dataSize = N * sizeof(float);
    double dataRate = dataSize / (elapsedTime*1e6);
    printf("Data transfer rate: %f GB/sec\n\n", dataRate);
        // уничтожение события 
    hipEventDestroy(start);
    hipEventDestroy(stop);

// e.Напишите программу, вычисляющую число пи методом интегрирования четверти окружности единичного радиуса
// (можно использовать формулу для вычисления площади четверти круга.В программе предусмотрите проверку на ошибку выполнения функции.-----------------------------------------
    printf("e) Write a program to calculate pi by integrating a quarter circle of unit radius\n");
    double* d_result, * h_result;
    double pi;

    // Выделение памяти на устройстве и хосте
    hipMalloc((void**)&d_result, N * sizeof(double));
    h_result = (double*)malloc(N * sizeof(double));

    // Запуск ядра
    //blockSize = 256;
    //numBlocks = (N + blockSize - 1) / blockSize;
    calculatePi << <numBlocks, blockSize >> > (d_result);

    // Копирование результатов на хост
    hipMemcpy(h_result, d_result, N * sizeof(double), hipMemcpyDeviceToHost);

    // Суммирование результатов
    pi = 0;
    for (int i = 0; i < N; i++) {
        pi += h_result[i];
    }

    // Умножаем на 4 для получения значения π
    pi *= 4;

    printf("Estimated value of pi: %f\n", pi);

    // Освобождение памяти
    free(h_result);
    hipFree(d_result);

    // Проверка на ошибки выполнения функции
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return 1;
    }


    return 0;
}
